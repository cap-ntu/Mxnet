#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"

namespace caffe {

int param_len = 0;
int **a;
int **a_host;
bool **b;
bool **b_host;

template <typename Dtype>
__global__ void SGDUpdate(int N, Dtype* g, Dtype* h,
    Dtype momentum, Dtype local_rate, float dvf_threshold, int* per_block_results, bool* whether_update) {
  __shared__ int sdata[CAFFE_CUDA_NUM_THREADS];
  sdata[threadIdx.x] = 0;
  __syncthreads();
  CUDA_KERNEL_LOOP(i, N) {
    // threshold works here
    // g[i] denotes the updates computed in this iteration
    // we first aggregated g[i] with h[i]
    // h[i] is the update computed in last iteration
    g[i] = h[i] = momentum*h[i] + local_rate*g[i];
    // ***********************************************************
    // How to offload following function to SmartNIC
    // MPI cannot be used anymore if we offload following function to SmartNIC
    // ***********************************************************
    if (g[i] >= 0 && g[i] <= dvf_threshold) {
      // here we drop g[i], store the dropped value in h[i]
      // this is different with the paper, since we use momentum SGD
      g[i] = 0;
      h[i] = 1.0/momentum * h[i]; // this is a trick for h[i], must use it
      // sdata is used to trace which update is useful
      sdata[threadIdx.x] += 1;
      whether_update[i] = false;
    } else if (g[i] < 0 && g[i] >= -dvf_threshold) {
      // here we dlso rop g[i], store the dropped value in h[i]
      g[i] = 0;
      h[i] = 1.0/momentum * h[i];
      sdata[threadIdx.x] += 1;
      whether_update[i] = false;
    } else {
      // here we do not drop g[i], store the dropped value in h[i]
      whether_update[i] = true;
    }
  }
  __syncthreads();
  for(int offset = blockDim.x/2; offset > 0; offset >>= 1) {  
    if(threadIdx.x < offset) {  
      // sdata is used to trace which update is useful
       sdata[threadIdx.x] += sdata[threadIdx.x + offset];  
    }  
    __syncthreads();  
  }  
  if(threadIdx.x == 0) {  
    per_block_results[blockIdx.x] = sdata[0];  
  }
}

template <typename Dtype>
long sgd_update_gpu(int N, Dtype* g, Dtype* h, Dtype momentum,
    Dtype local_rate, float dvf_threshold, int* update_vector, int param_id) {
  long dn = 0;
  size_t block_num = CAFFE_GET_BLOCKS(N);
  int *d_partial_sums = 0;
  int *d_partial_sums_host = 0;
  bool *whether_update = 0;
  bool *whether_update_host = 0;
  if (param_id == param_len) {
    param_len += 1;
    hipMalloc((void**)&d_partial_sums, sizeof(int) * block_num);
    d_partial_sums_host = (int*) malloc(sizeof(int) * block_num);
    hipMalloc((void**)&whether_update, sizeof(bool) * N);
    whether_update_host = (bool*) malloc(sizeof(bool) * N);

    int **a_tmp = (int**)malloc(sizeof(int*) * param_len);
    int **a_host_tmp = (int**)malloc(sizeof(int*) * param_len);
    bool **b_tmp = (bool**)malloc(sizeof(bool*) * param_len);
    bool **b_host_tmp = (bool**)malloc(sizeof(bool*) * param_len);

    for (int i=0; i<(param_len-1); i++) {
      a_tmp[i] = a[i];
      a_host_tmp[i] = a_host[i];
      b_tmp[i] = b[i];
      b_host_tmp[i] = b_host[i];
    }
    a_tmp[param_len-1] = d_partial_sums;
    a_host_tmp[param_len-1] = d_partial_sums_host;
    b_tmp[param_len-1] = whether_update;
    b_host_tmp[param_len-1] = whether_update_host;
    free(a);
    free(a_host);
    free(b);
    free(b_host);
    a = a_tmp;
    a_host = a_host_tmp;
    b = b_tmp;
    b_host = b_host_tmp;
  } else {
    d_partial_sums = a[param_id];
    d_partial_sums_host = a_host[param_id];
    whether_update = b[param_id];
    whether_update_host = b_host[param_id];
  }

  SGDUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<block_num, CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, momentum, local_rate, dvf_threshold, d_partial_sums, whether_update);
  CUDA_POST_KERNEL_CHECK;

  //**************************************************************
  // Communication happends here, delete the MPI related communication function
  // need to implemente the communication  function using smart NIC APIs
  // following is what MPI do in previous implementation:
  //    MPI_Send g to rank 0, in Rank 0, g[i] += receviced_g[i]
  //    Then, in Rank0, average the update: g[i] = g[i] / RankNumber
  //    MPI_Bcast g to other ranks
  // g would be used to update parameteres in another function, and we would not care it.
  //*************************************************************

  hipMemcpy(d_partial_sums_host, d_partial_sums, sizeof(int)*block_num, hipMemcpyDeviceToHost);
  hipMemcpy(whether_update_host, whether_update, sizeof(bool)*N, hipMemcpyDeviceToHost);

  for (int i=0; i < block_num; i++) {
    dn += d_partial_sums_host[i];
  } 
  for (int i=0; i < N; i++) {
    update_vector[i] += whether_update_host[i];
  }

  // hipFree(d_partial_sums);
  // hipFree(whether_update);
  // free(d_partial_sums_host);
  // free(whether_update_host);
  return dn;
}
template long sgd_update_gpu<float>(int, float*, float*, float, float, float, int*, int);
template long sgd_update_gpu<double>(int, double*, double*, double, double, float, int*, int);

}  // namespace caffe
